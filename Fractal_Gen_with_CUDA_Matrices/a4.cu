#include "hip/hip_runtime.h"
/*
 * a4.cu
 * Assignment 4 - COSC330
 * Author: Christian Caburian
 *
 * Implements a parallel multithreaded version of the Mandelbrot set
 * program in the CUDA environment.
 * User can specify width and height of the image.
 *
 * Usage: a4 <width> <height>
 *
 * Compile: make
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bmpfile.h"
/**
  * Parses console arguments
  * argc: no. of args
  * argv: array of args
  * width and height: locations to store width and height of the image
  */
__host__ int
parse_args(int argc,  char *argv[], double *width, double *height) 
{
    if ((argc != 3) ||
        ((*width = atoi(argv[1])) <= 0) ||
        ((*height = atoi(argv[2])) <= 0)) {

        fprintf(stderr, "Usage: %s width height\n", argv[0]);
        return(-1);
    }
    return 0;
}
/**
  * Copies memory between host and device
  * src: source to be copied
  * dst: destination to copy to
  * size: size of memory to be copied
  * kind: copy direction; from host to device or vice versa
  * err: location of hipError_t variable
  * msg: announces operation; reports error
  */
__host__ void
cudaMemcp_check_err(double *src,
                    double *dst,
                    size_t size,
                    enum hipMemcpyKind kind,
                    hipError_t *err,
                    const char *msg)
{
    printf("Copying %s\n", msg);
    *err = hipMemcpy(src, dst, size, kind);
    if (*err != hipSuccess) {
        fprintf(stderr, "Failed to copy %s (error code %s)!\n",
                msg, hipGetErrorString(*err));
        exit(EXIT_FAILURE);
    }
}
/**
  * Frees CUDA memory, with error checking
  * dev_mat: device matrix to free
  * err: location of hipError_t variable
  */
__host__ void
cudaFree_check_err(double *dev_mat, hipError_t *err)
{
    *err = hipFree(dev_mat);
    if (*err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
                hipGetErrorString(*err));
        exit(EXIT_FAILURE);
    }
} 
/**
  * Initializes the pixel reference matrices for potential (x, y) coordinates.
  * x and y: flattened 2D matrices of potential x and y coordinates in the
  * 		 Mandelbrot set.
  * width and height: dimensions of the image.
  */
__host__ void
pixel_ref(double *x, double *y, double width, double height)
{
    // Mandelbrot values
    double resolution = 8700.0;
    double xcenter = -0.55;
    double ycenter = 0.60;

    int i, col, row;
    int xoffset = -(width - 1) /2;
    int yoffset = (height -1) / 2;

    for(col = 0, i = 0; col < width; ++col) {
        for(row = 0; row < height; ++row, ++i) {
            x[i] = xcenter + (xoffset + col) / resolution;
            y[i] = ycenter + (yoffset - row) / resolution;
        }
    }
}
/**
  * Computes the Mandelbrot pixel value
  * x and y: flattened 2D matrices of potential x and y coordinates in the
  * 		 Mandelbrot set.
  * pixel_mat: Result/output matrix of Mandelbrot pixel values
  * N: Number of pixels in the image; width * height
  */
__global__ void
mandelbrot(const double *x, const double *y, double *pixel_mat, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N) {

        // Colour values
        double colour_max = 240.0;
        double gradient_colour_max = 230.0;

        //Mandelbrot stuff
        double a = 0;
        double b = 0;
        double aold = 0;
        double bold = 0;
        double zmagsqr = 0;
        int iter = 0;
        int max_iter = 1000;
        float iter_ratio;

        //Check if the x,y coordinates are part of the Mandelbrot set
        while(iter < 1000 && zmagsqr <= 4.0) {
        
            ++iter;
            a = (aold * aold) - (bold * bold) + x[i];
            b = 2.0 * aold*bold + y[i];

            zmagsqr = a*a + b*b;

            aold = a;
            bold = b;
        }

        // Generate the colour of the pixel from iter value
        iter_ratio = (float)iter / (float)max_iter;
        pixel_mat[i] = colour_max - iter_ratio * gradient_colour_max;
    }
}
/**
  * Computes the colour gradient
  * colour: the output vector
  * x: the gradient (beetween 0 and 360)
  * min and max: variation of the RGB channels (Move3D 0 -> 1)
  * Check wiki for more details on the colour science: en.wikipedia.org/wiki/HSL_and_HSV
  */
__host__ void
ground_colour_mix(double* colour, double x, double min, double max)
{
 /*
  * Red = colour[0]
  * Green = colour[1]
  * Blue = colour[2]
  */
    double posSlope = (max-min)/60;
    double negSlope = (min-max)/60;

    if( x < 60 ) {
        colour[0] = max;
        colour[1] = posSlope*x+min;
        colour[2] = min;
        return;
    }
    else if ( x < 120 ) {
        colour[0] = negSlope*x+2.0*max+min;
        colour[1] = max;
        colour[2] = min;
        return;
    }
    else if ( x < 180  ) {
        colour[0] = min;
        colour[1] = max;
        colour[2] = posSlope*x-2.0*max+min;
        return;
    }
    else if ( x < 240  ) {
        colour[0] = min;
        colour[1] = negSlope*x+4.0*max+min;
        colour[2] = max;
        return;
    }
    else if ( x < 300  ) {
        colour[0] = posSlope*x-4.0*max+min;
        colour[1] = min;
        colour[2] = max;
        return;
    }
    else {
        colour[0] = max;
        colour[1] = min;
        colour[2] = negSlope*x+6*max;
        return;
    }
}
/**
  * Converts a matrix of pixels to a BMP file
  * pixel_mat: matrix of pixels.
  * width and height: dimensions of the image.
  */
__host__ void
pixel_mat_to_bmp(double *pixel_mat, double width, double height)
{
    const char *filename = "my_mandelbrot_fractal.bmp";
    bmpfile_t *bmp;
    rgb_pixel_t pixel = {0, 0, 0, 0};
    double colour[3];
    double colour_depth = 255.0;

    // Create BMP
    printf("Writing result to BMP file...\n");
    if ((bmp = bmp_create(width, height, 32)) == NULL) {
        fprintf(stderr, "Failed to create BMP!\n");
        exit(EXIT_FAILURE);
    }

    // Convert pixel matrix to BMP file
    int col, row;
    for(col = 0; col < width; ++col){
        for(row = 0; row < height; ++row){

            ground_colour_mix(colour,
                              pixel_mat[col*(int)height+row],
                              1,
                              colour_depth);
                              
            pixel.red = colour[0];
            pixel.green = colour[1];
            pixel.blue = colour[2];
            bmp_set_pixel(bmp, col, row, pixel);
        }
    }

    // Save BMP file
    if (!bmp_save(bmp, filename)) {
        fprintf(stderr, "Failed to save BMP!\n");
        exit(EXIT_FAILURE);
    }
    bmp_destroy(bmp);
    printf("File saved.\n");
}
/* Mandelbrot Set Image Demonstration
 *
 * This is a simple parallel multithread implementation
 * that computes a Mandelbrot set and produces a corresponding
 * Bitmap image. The program demonstrates the use of a colour
 * gradient
 *
 * This program uses the algorithm outlined in:
 *   "Building Parallel Programs: SMPs, Clusters And Java", Alan Kaminsky
 *
 * This program requires libbmp for all bitmap operations.
 *
 */
int
main(int argc, char *argv[])
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Image size to be used
    double width, height;
    if (parse_args(argc, argv, &width, &height) < 0) exit(EXIT_FAILURE);
    printf("Width: %.01f, Height: %.01f\n", width, height);
    int N = (int)width*(int)height;
    size_t size = N * sizeof(double);

    // Allocate memory and verify the allocations succeeded
    // Host matrices
    double *x = (double *)malloc(size);
    double *y = (double *)malloc(size);
    double *pixel_mat = (double *)malloc(size);
    if (pixel_mat == NULL || x == NULL || y == NULL) {
        fprintf(stderr, "Failed to allocate host pixel_mat!\n");
        exit(EXIT_FAILURE);
    }
    // Device matrices
    double *d_x = NULL;
    err = hipMalloc((void **)&d_x, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device x (error code %s)!\n", 
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    double *d_y = NULL;
    err = hipMalloc((void **)&d_y, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device y (error code %s)!\n", 
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    double *d_pixel_mat = NULL;
    err = hipMalloc((void **)&d_pixel_mat, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device pixel_mat (error code %s)!\n", 
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Determine where in the Mandelbrot set, the pixel is referencing
    pixel_ref(x, y, width, height);

    // Copy the pixel reference host matrices x and y to the device
    cudaMemcp_check_err(d_x, x, size, hipMemcpyHostToDevice, &err, 
                        "matrix X from host to device...");
    cudaMemcp_check_err(d_y, y, size, hipMemcpyHostToDevice, &err, 
                        "matrix Y from host to device...");

    // Launch kernel to determine Mandelbrot set
    int threadsPerBlock = 256;
    int blocksPerGrid = (N+threadsPerBlock-1)/threadsPerBlock;
    printf("Calling kernel with threadsPerBlock = %d, blocksPerGrid = %d...\n",
            threadsPerBlock, blocksPerGrid);
    mandelbrot<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_pixel_mat, N);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch Mandelbrot kernel (error code %s)!\n", 
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result matrix in device memory to the host result matrix
    // in host memory.
    cudaMemcp_check_err(pixel_mat, d_pixel_mat, size, 
                        hipMemcpyDeviceToHost, &err, 
                        "result from device to host...");

    // Write the pixel_mat to a BMP file
    pixel_mat_to_bmp(pixel_mat, width, height);

    // Free device global memory
    cudaFree_check_err(d_x, &err);
    cudaFree_check_err(d_y, &err);
    cudaFree_check_err(d_pixel_mat, &err);

    // Free host memory
    free(pixel_mat);
    free(x);
    free(y);

    // Reset the device and exit
    err = hipDeviceReset();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", 
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 0;
}
